#include "hip/hip_runtime.h"
#include <math.h>
#include <omp.h>
#include <mpi.h>
#include <pthread.h>
#include <cassert>

#include "classifier.h"
#include "util.h"


/** SECTION: Constants and hyperparameters **/
#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))

#define NUM_PARAMETER (OFFSET21 + 4)

static int mpi_size, mpi_rank;
static char processor_name[MPI_MAX_PROCESSOR_NAME];
static int iam_root;


/** SECTION: GPU manipulation **/
#define NGPU    4
static hipStream_t streams[NGPU];
static float *a_input_gpu[NGPU], *a_conv1_gpu[NGPU];
static float *a_layernorm1_gpu[NGPU];
static float *a_pool1_gpu[NGPU], *a_conv2_gpu[NGPU];
static float *a_pool2_gpu[NGPU], *a_conv3_gpu[NGPU];
static float *a_conv4_gpu[NGPU];
static float *a_conv5_gpu[NGPU];
static float *a_conv6_gpu[NGPU];
static float *a_layernorm6_gpu[NGPU];
static float *a_collapse_gpu[NGPU];
static float *a_linear1_gpu[NGPU];
static float *a_linear2_gpu[NGPU];
static float *a_linear3_gpu[NGPU];

static float *w_conv1_gpu[NGPU], *b_conv1_gpu[NGPU];
static float *w_conv2_gpu[NGPU], *b_conv2_gpu[NGPU];
static float *w_conv3_gpu[NGPU], *b_conv3_gpu[NGPU];
static float *w_conv4_gpu[NGPU], *b_conv4_gpu[NGPU];
static float *w_conv5_gpu[NGPU], *b_conv5_gpu[NGPU];
static float *w_conv6_gpu[NGPU], *b_conv6_gpu[NGPU];
static float *w_fc1_gpu[NGPU], *b_fc1_gpu[NGPU];
static float *w_fc2_gpu[NGPU], *b_fc2_gpu[NGPU];
static float *w_fc3_gpu[NGPU], *b_fc3_gpu[NGPU];
static float *gamma_conv1_gpu[NGPU], *beta_conv1_gpu[NGPU];
static float *gamma_conv6_gpu[NGPU], *beta_conv6_gpu[NGPU];


/** SECTION: DEBUGGING **/
#define DEBUG 0
#if DEBUG == 1
#define DEBUG_PRINT(...) do { \
  printf("(%s|rank=%d) ", processor_name, mpi_rank); \
  printf(__VA_ARGS__); \
} while (0)
#else
#define DEBUG_PRINT(...)
#endif

int checksum(float *buf, int N) {
  int sum = 0;
  for (int i = 0; i < N; ++i)
    sum += (int) buf[i];

  return sum;
}


/** SECTION: Hyperparams **/
#define ROOT_INPUT_N    (2048 + 256 * 3)
#define NONROOT_INPUT_N (2048 - 256)

#define POP_BATCH_SIZE 16
#define COMPUTE_BATCH_SIZE 4

#define C1D_K3_BM 16
#define C1D_K3_BN 8
#define C1D_K3_BK 8

#define C1D_K7_BM 8
#define C1D_K7_BN 18
#define C1D_K7_BK 4

#define LIN_NAIVE_BM 4
#define LIN_NAIVE_BN 16

#define LIN_REG_BM 4
#define LIN_REG_BN 16
#define LIN_REG_BK 32

#define LNORM_CHAN 256  // FIXED
#define LNORM_CHPT 2
#define LNORM_TPB  (LNORM_CHAN/LNORM_CHPT)  // FIXED


/** SECTION: Tensor **/

// Multi-dimensional matrix containing fp32 elements
struct Tensor {
  Tensor(std::vector<int> shape_);
  Tensor(std::vector<int> shape_, float *buf_);
  ~Tensor();
  int num_elem();
  void fill_zeros();

  float *buf = nullptr;
  int ndim = 0;
  int shape[4];
};

Tensor::Tensor(std::vector<int> shape_) {
  ndim = shape_.size();
  for (int i = 0; i < ndim; ++i) { shape[i] = shape_[i]; }
  int N_ = num_elem();
  buf = (float *) calloc(N_, sizeof(float));
}

Tensor::Tensor(std::vector<int> shape_, float *buf_) {
  ndim = shape_.size();
  for (int i = 0; i < ndim; ++i) { shape[i] = shape_[i]; }
  int N_ = num_elem();
  buf = (float *) calloc(N_, sizeof(float));
  for (int n = 0; n < N_; ++n) { buf[n] = buf_[n]; }
}

Tensor::~Tensor() {
  if (buf != nullptr) free(buf);
}

int Tensor::num_elem() {
  int sz = 1;
  for (int i = 0; i < ndim; ++i) { sz *= shape[i]; }
  return sz;
}

void Tensor::fill_zeros() {
  int N_ = num_elem();
  for (int n = 0; n < N_; ++n) { buf[n] = 0.0; }
}

/** SECTION: Kernels **/

static __global__ void conv1d_k3_cuda(
  float *input, float *weight, float *bias, float *output,
  int num_batch, int len_output, int in_channels, int out_channels,
  int relu
) {
  /** PARAMS **/
  // input: float[batch_size, in_channels, len_input]
  // weight: float[out_channels, in_channels, kernel_size]
  // bias: float[out_channels]
  // output: float[batch_size, out_channels, len_output]

  /** CONSTS **/
  const int BB = COMPUTE_BATCH_SIZE;
  const int BM = C1D_K3_BM;
  const int BN = C1D_K3_BN;
  const int BK = C1D_K3_BK;

  const int KERNEL_SIZE = 3;
  const int len_input = len_output + KERNEL_SIZE - 1;
  const int single_input_size = in_channels * len_input;
  const int single_output_size = out_channels * len_output;

  /** ASSERTION **/
  #if DEBUG == 1
  if (BM * BN < BM * BK * KERNEL_SIZE) {
    if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
      // printf("conv1d_k3_cuda: num of threads are insufficient for kernel load!\n");
      // return;
    }
  }
  #endif
  
  /** VARS **/
  float val[BB] = {0.0f};
  
  // output block
  int oblock_m_offset = blockIdx.x * BM;
  int oblock_n_offset = blockIdx.y * BN;
  int len_oblock_m = min(BM, out_channels - oblock_m_offset);
  int len_oblock_n = min(BN, len_output - oblock_n_offset);
  int othread_m_offset = threadIdx.x / len_oblock_n;
  int othread_n_offset = threadIdx.x % len_oblock_n;

  int othread_valid = othread_m_offset < len_oblock_m;
  
  /** SMEM **/
  __shared__ float input_buf[BB][BK][BN + KERNEL_SIZE - 1 + 4];
  __shared__ float weight_buf[BM][BK][KERNEL_SIZE + 4];

  /** LOOP OVER K **/
  for (int bk = 0; bk < in_channels; bk += BK) {
    // load input
    int iblock_k_offset = bk;
    int iblock_n_offset = oblock_n_offset;
    int len_iblock_k = min(BK, in_channels - iblock_k_offset);
    int len_iblock_n = min(BN + KERNEL_SIZE - 1, len_input - iblock_n_offset);
    int ithread_k_offset = threadIdx.x / len_iblock_n;
    int ithread_n_offset = threadIdx.x % len_iblock_n;

    int ithread_valid = ithread_k_offset < len_iblock_k;

    if (ithread_valid) {
      for (int bb = 0; bb < num_batch; bb++) {
        input_buf[bb][ithread_k_offset][ithread_n_offset] = input[
          bb * single_input_size
           + (iblock_k_offset + ithread_k_offset) * len_input
           + iblock_n_offset + ithread_n_offset
        ];
      }
    }

    // load weight
    int wblock_m_offset = oblock_m_offset;
    int wblock_k_offset = bk;
    int len_wblock_m = min(BM, out_channels - wblock_m_offset);
    int len_wblock_k = min(BK, in_channels - wblock_k_offset);
    int wthread_m_offset = threadIdx.x / len_wblock_k;
    int wthread_k_offset = threadIdx.x % len_wblock_k;

    int wthread_valid = wthread_m_offset < len_wblock_m;

    if (wthread_valid) {
      for (int ks = 0; ks < KERNEL_SIZE; ks++) {
        weight_buf[wthread_m_offset][wthread_k_offset][ks] = weight[
          (wblock_m_offset + wthread_m_offset) * in_channels * KERNEL_SIZE
          + (wblock_k_offset + wthread_k_offset) * KERNEL_SIZE + ks
        ];
      }
    }

    __syncthreads();

    // compute
    if (othread_valid) {
      for (int bb = 0; bb < num_batch; bb++) {
        for (int k = 0; k < BK; k++) {
          for (int ks = 0; ks < KERNEL_SIZE; ks++) {
            val[bb] += weight_buf[othread_m_offset][k][ks] * input_buf[bb][k][othread_n_offset + ks];
          }
        }
      }
    }

    __syncthreads();
  }

  /** STORE **/
  if (othread_valid) {
    for (int bb = 0; bb < num_batch; bb++) {
      val[bb] += bias[oblock_m_offset + othread_m_offset];
      if (relu && val[bb] < 0.0f) val[bb] = 0.0f;
      output[
        bb * single_output_size
         + (oblock_m_offset + othread_m_offset) * len_output
         + oblock_n_offset + othread_n_offset
      ] = val[bb];
    }
  }
}

static __global__ void conv1d_k7_cuda(
  float *input, float *weight, float *bias, float *output,
  int num_batch, int len_output, int in_channels, int out_channels,
  int relu, int mpool3=0, float *pooled_output=nullptr
) {
  /** PARAMS **/
  // input: float[batch_size, in_channels, len_input]
  // weight: float[out_channels, in_channels, kernel_size]
  // bias: float[out_channels]
  // output: float[batch_size, out_channels, len_output]

  /** CONSTS **/
  const int BB = COMPUTE_BATCH_SIZE;
  const int BM = C1D_K7_BM;
  const int BN = C1D_K7_BN;
  const int BK = C1D_K7_BK;

  const int KERNEL_SIZE = 7;
  const int len_input = len_output + KERNEL_SIZE - 1;

  const int single_input_size = in_channels * len_input;
  const int single_output_size = out_channels * len_output;

  /** ASSERTION **/
  #if DEBUG == 1
  if (BM * BN < BM * BK * KERNEL_SIZE) {
    if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
      // printf("conv1d_k3_cuda: num of threads are insufficient for kernel load!\n");
      // return;
    }
  }
  #endif
  
  /** VARS **/
  float val[BB] = {0.0f};
  
  // output block
  int oblock_m_offset = blockIdx.x * BM;
  int oblock_n_offset = blockIdx.y * BN;
  int len_oblock_m = min(BM, out_channels - oblock_m_offset);
  int len_oblock_n = min(BN, len_output - oblock_n_offset);
  int othread_m_offset = threadIdx.x / len_oblock_n;
  int othread_n_offset = threadIdx.x % len_oblock_n;

  int othread_valid = othread_m_offset < len_oblock_m;
  
  /** SMEM **/
  __shared__ float input_buf[BB][BK][BN + KERNEL_SIZE - 1 + 4];
  __shared__ float weight_buf[BM][BK][KERNEL_SIZE + 4];

  /** LOOP OVER K **/
  for (int bk = 0; bk < in_channels; bk += BK) {
    // load input
    int iblock_k_offset = bk;
    int iblock_n_offset = oblock_n_offset;
    int len_iblock_k = min(BK, in_channels - iblock_k_offset);
    int len_iblock_n = min(BN + KERNEL_SIZE - 1, len_input - iblock_n_offset);
    int ithread_k_offset = threadIdx.x / len_iblock_n;
    int ithread_n_offset = threadIdx.x % len_iblock_n;

    int ithread_valid = ithread_k_offset < len_iblock_k;

    if (ithread_valid) {
      for (int bb = 0; bb < num_batch; bb++) {
        input_buf[bb][ithread_k_offset][ithread_n_offset] = input[
          bb * single_input_size
           + (iblock_k_offset + ithread_k_offset) * len_input
           + iblock_n_offset + ithread_n_offset
        ];
      }
    }

    // load weight
    int wblock_m_offset = oblock_m_offset;
    int wblock_k_offset = bk;
    int len_wblock_m = min(BM, out_channels - wblock_m_offset);
    int len_wblock_k = min(BK, in_channels - wblock_k_offset);
    int wthread_m_offset = threadIdx.x / len_wblock_k;
    int wthread_k_offset = threadIdx.x % len_wblock_k;

    int wthread_valid = wthread_m_offset < len_wblock_m;

    if (wthread_valid) {
      for (int ks = 0; ks < KERNEL_SIZE; ks++) {
        weight_buf[wthread_m_offset][wthread_k_offset][ks] = weight[
          (wblock_m_offset + wthread_m_offset) * in_channels * KERNEL_SIZE
          + (wblock_k_offset + wthread_k_offset) * KERNEL_SIZE + ks
        ];
      }
    }

    __syncthreads();

    // compute
    if (othread_valid) {
      for (int bb = 0; bb < num_batch; bb++) {
        for (int k = 0; k < BK; k++) {
          for (int ks = 0; ks < KERNEL_SIZE; ks++) {
            val[bb] += weight_buf[othread_m_offset][k][ks] * input_buf[bb][k][othread_n_offset + ks];
          }
        }
      }
    }

    __syncthreads();
  }

  /** STORE **/
  if (othread_valid) {
    for (int bb = 0; bb < num_batch; bb++) {
      val[bb] += bias[oblock_m_offset + othread_m_offset];
      if (relu && val[bb] < 0.0f) val[bb] = 0.0f;
      output[
        bb * single_output_size
         + (oblock_m_offset + othread_m_offset) * len_output
         + oblock_n_offset + othread_n_offset
      ] = val[bb];
    }
  }
}

static __global__ void linear_naive_cuda(
  float *input, float *weight, float *bias, float *output,
  int num_batch, int in_channels, int out_channels,
  int relu
) {
  /** PARAMS **/
  // input: float[batch_size, in_channels]
  // weight: float[out_channels, in_channels]
  // bias: float[out_channels]
  // output: float[batch_size, out_channels]

  /** CONSTS **/
  const int BM = LIN_NAIVE_BM;
  const int BN = LIN_NAIVE_BN;

  /** VARS **/
  int block_batch_idx = blockIdx.x * BM;
  int block_outchan_idx = blockIdx.y * BN;
  int block_batch_len = min(BM, num_batch - block_batch_idx);
  int block_outchan_len = min(BN, out_channels - block_outchan_idx);

  int thread_batch_idx = block_batch_idx + threadIdx.x;
  int thread_outchan_idx = block_outchan_idx + threadIdx.y;

  if (thread_batch_idx < num_batch && thread_outchan_idx < out_channels) {
    /** COMPUTE **/
    float val = 0.0f;
    for (int k = 0; k < in_channels; k++) {
      val += weight[thread_outchan_idx * in_channels + k] * input[thread_batch_idx * in_channels + k];
    }

    /** STORE **/
    val += bias[thread_outchan_idx];
    if (relu && val < 0.0f) val = 0.0f;
    output[thread_batch_idx * out_channels + thread_outchan_idx] = val;
  }
}

static __global__ void linear_reg_cuda(
  float *input, float *weight, float *bias, float *output,
  int num_batch, int in_channels, int out_channels,
  int relu
) {
  /** PARAMS **/
  // input: float[batch_size, in_channels]
  // weight: float[out_channels, in_channels]
  // bias: float[out_channels]
  // output: float[batch_size, out_channels]

  /** CONSTS **/
  const int BM = LIN_REG_BM;
  const int BN = LIN_REG_BN;
  const int BK = LIN_REG_BK;
  const int LDPT_INPUT = BK / BN;
  const int LDPT_WEIGHT = BK / BM;

  /** VARS **/
  float val = 0.0f;

  int oblock_m = blockIdx.x * BM;
  int oblock_n = blockIdx.y * BN;

  /** SMEM **/
  __shared__ float input_buf[BM][BK];
  __shared__ float weight_buf[BK][BN];

  /** LOOP OVER K **/
  for (int bk = 0; bk < in_channels; bk += BK) {
    // load input
    for (int ld_input = 0; ld_input < LDPT_INPUT; ld_input++) {
      input_buf[threadIdx.x][threadIdx.y * LDPT_INPUT + ld_input] = input[
        in_channels * (oblock_m + threadIdx.x)
        + bk + threadIdx.y * LDPT_INPUT + ld_input
      ];
    }

    // load weight
    for (int ld_weight = 0; ld_weight < LDPT_WEIGHT; ld_weight++) {
      weight_buf[threadIdx.x * LDPT_WEIGHT + ld_weight][threadIdx.y] = weight[
        in_channels * (oblock_n + threadIdx.y)
        + bk + threadIdx.x * LDPT_WEIGHT + ld_weight
      ];
    }

    __syncthreads();

    // compute
    for (int k = 0; k < BK; k++) {
      val += weight_buf[k][threadIdx.y] * input_buf[threadIdx.x][k];
    }

    __syncthreads();
  }

  /** STORE **/
  val += bias[oblock_n + threadIdx.y];
  if (relu && val < 0.0f) val = 0.0f;
  output[out_channels * (oblock_m + threadIdx.x) + oblock_n + threadIdx.y] = val;

}

static __global__ void layernorm_cuda(
  float *input, float *gamma, float *beta, float *output,
  int num_batch, int num_channels, int len_input
) {

  int now_batch = blockIdx.x;
  int single_input_size = num_channels * len_input;
  int single_output_size = num_channels * len_input;

  int thread_stride = LNORM_CHPT * len_input;

  __shared__ float sum1[LNORM_TPB];
  __shared__ float sum2[LNORM_TPB];

  float psum1 = 0.0f, psum2 = 0.0f;
  for (int i = 0; i < thread_stride; ++i) {
    psum1 += input[now_batch * single_input_size + threadIdx.x * thread_stride + i];
    psum2 += input[now_batch * single_input_size + threadIdx.x * thread_stride + i]
      * input[now_batch * single_input_size + threadIdx.x * thread_stride + i];
  }

  __syncthreads();

  sum1[threadIdx.x] = psum1;
  sum2[threadIdx.x] = psum2;
  
  if (threadIdx.x == 0) {
    for (int i = 1; i < LNORM_TPB; i++) {
      sum1[0] += sum1[i];
      sum2[0] += sum2[i];
    }
  }

  __syncthreads();

  float mean1 = sum1[0] / (float)single_input_size;
  float mean2 = sum2[0] / (float)single_input_size;

  float var = mean2 - mean1 * mean1;

  for (int i = 0; i < thread_stride; ++i) {
    output[now_batch * single_output_size + threadIdx.x * thread_stride + i]
    = (input[now_batch * single_input_size + threadIdx.x * thread_stride + i] - mean1)
    / sqrtf(var + 1e-5) * gamma[threadIdx.x * thread_stride + i] + beta[threadIdx.x * thread_stride + i];
  }
  
}

static __global__ void maxpool1d_k3_cuda(
  float *input, float *output,
  int num_batch, int num_channels, int len_input,
  int relu
) {
  int POOL_SIZE = 3;
  int single_input_size = num_channels * len_input;
  int single_output_size = num_channels * (len_input / POOL_SIZE);
  int len_output = len_input / POOL_SIZE;

  int now_batch = blockIdx.x;
  int now_ol = threadIdx.x;
  for (int oc = 0; oc < num_channels; ++oc) {
    float mx = -1e99;
    for (int ks = 0; ks < POOL_SIZE; ++ks) {
      float val = input[now_batch * single_input_size + oc * len_input + ks + now_ol * POOL_SIZE];
      if (val > mx) mx = val;
    }
    if (relu && mx < 0.0f) mx = 0.0f;
    output[now_batch * single_output_size + oc * len_output + now_ol] = mx;
  }

}


/** SECTION: COMPUTE_ENGINE **/

struct ComputeEngine {
public:
  ComputeEngine(float *parameter, int num_input, int gpu_idx);
  ~ComputeEngine();

  void set_input(float *input_buf_);
  void set_output(float *output_buf_);

  void run();
  void join();

  void push(int num_input);

private:
  // Input
  float *input_buf;
  float *output_buf;
  int num_input;
  
  // GPU
  int gpu_idx;

  // Queue
  float *input_to_process;
  int num_input_ready;
  int num_input_processed;
  pthread_mutex_t mutex_queue;
  pthread_cond_t cond_queue;

  int pop();
  void inference(int num_input);

  // Runner
  static void *run_func(void *arg);
  pthread_t thread;

  // Parameters
  Tensor *w_fc1, *w_fc2, *w_fc3,
      *b_fc1, *b_fc2, *b_fc3, *gamma_conv1, *beta_conv1, *gamma_conv6, *beta_conv6;

  // Activations
  Tensor *a_relu2, *a_pool2;
  Tensor *a_conv6, *a_layernorm6, *a_relu6, *a_pool6;
  Tensor *a_linear3;
};

ComputeEngine *compute_engines[NGPU];

ComputeEngine::ComputeEngine(float *parameter_, int num_input_, int gpu_idx_) {
  // Initialize member variables
  input_buf = nullptr;
  output_buf = nullptr;
  num_input = num_input_;

  gpu_idx = gpu_idx_;

  // Initialize queue
  num_input_ready = 0;
  num_input_processed = 0;
  pthread_mutex_init(&mutex_queue, NULL);
  pthread_cond_init(&cond_queue, NULL);

  // Initialize CUDA
  for (int i = 0; i < NGPU; ++i) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamCreate(&streams[i]));
  }

  // Initialize parameters
  for (int i = 0; i < NGPU; ++i) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMalloc(&a_input_gpu[i], COMPUTE_BATCH_SIZE * 70 * 1014 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_conv1_gpu[i], COMPUTE_BATCH_SIZE * 256 * 1008 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_layernorm1_gpu[i], COMPUTE_BATCH_SIZE * 256 * 1008 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_pool1_gpu[i], COMPUTE_BATCH_SIZE * 256 * 336 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_conv2_gpu[i], COMPUTE_BATCH_SIZE * 256 * 330 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_pool2_gpu[i], COMPUTE_BATCH_SIZE * 256 * 110 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_conv3_gpu[i], COMPUTE_BATCH_SIZE * 256 * 108 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_conv4_gpu[i], COMPUTE_BATCH_SIZE * 256 * 106 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_conv5_gpu[i], COMPUTE_BATCH_SIZE * 256 * 104 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_conv6_gpu[i], COMPUTE_BATCH_SIZE * 256 * 102 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_layernorm6_gpu[i], COMPUTE_BATCH_SIZE * 256 * 102 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_collapse_gpu[i], COMPUTE_BATCH_SIZE * 8704 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_linear1_gpu[i], COMPUTE_BATCH_SIZE * 1024 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_linear2_gpu[i], COMPUTE_BATCH_SIZE * 1024 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&a_linear3_gpu[i], COMPUTE_BATCH_SIZE * 4 * sizeof(float)));
    
    CHECK_CUDA(hipMalloc(&w_conv1_gpu[i], 256 * 70 * 7 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&b_conv1_gpu[i], 256 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&w_conv2_gpu[i], 256 * 256 * 7 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&b_conv2_gpu[i], 256 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&w_conv3_gpu[i], 256 * 256 * 3 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&b_conv3_gpu[i], 256 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&w_conv4_gpu[i], 256 * 256 * 3 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&b_conv4_gpu[i], 256 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&w_conv5_gpu[i], 256 * 256 * 3 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&b_conv5_gpu[i], 256 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&w_conv6_gpu[i], 256 * 256 * 3 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&b_conv6_gpu[i], 256 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&w_fc1_gpu[i], 1024 * 8704 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&b_fc1_gpu[i], 1024 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&w_fc2_gpu[i], 1024 * 1024 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&b_fc2_gpu[i], 1024 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&w_fc3_gpu[i], 4 * 1024 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&b_fc3_gpu[i], 4 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&gamma_conv1_gpu[i], 256 * 1008 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&beta_conv1_gpu[i], 256 * 1008 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&gamma_conv6_gpu[i], 256 * 102 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&beta_conv6_gpu[i], 256 * 102 * sizeof(float)));

    CHECK_CUDA(hipMemcpyAsync(
      w_conv1_gpu[i], parameter_ + OFFSET0, 256 * 70 * 7 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      b_conv1_gpu[i], parameter_ + OFFSET1, 256 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      gamma_conv1_gpu[i], parameter_ + OFFSET2, 256 * 1008 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      beta_conv1_gpu[i], parameter_ + OFFSET3, 256 * 1008 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      w_conv2_gpu[i], parameter_ + OFFSET4, 256 * 256 * 7 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      b_conv2_gpu[i], parameter_ + OFFSET5, 256 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      w_conv3_gpu[i], parameter_ + OFFSET6, 256 * 256 * 3 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      b_conv3_gpu[i], parameter_ + OFFSET7, 256 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      w_conv4_gpu[i], parameter_ + OFFSET8, 256 * 256 * 3 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      b_conv4_gpu[i], parameter_ + OFFSET9, 256 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      w_conv5_gpu[i], parameter_ + OFFSET10, 256 * 256 * 3 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      b_conv5_gpu[i], parameter_ + OFFSET11, 256 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      w_conv6_gpu[i], parameter_ + OFFSET12, 256 * 256 * 3 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      b_conv6_gpu[i], parameter_ + OFFSET13, 256 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      gamma_conv6_gpu[i], parameter_ + OFFSET14, 256 * 102 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      beta_conv6_gpu[i], parameter_ + OFFSET15, 256 * 102 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      w_fc1_gpu[i], parameter_ + OFFSET16, 1024 * 8704 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      b_fc1_gpu[i], parameter_ + OFFSET17, 1024 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      w_fc2_gpu[i], parameter_ + OFFSET18, 1024 * 1024 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      b_fc2_gpu[i], parameter_ + OFFSET19, 1024 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      w_fc3_gpu[i], parameter_ + OFFSET20, 4 * 1024 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    CHECK_CUDA(hipMemcpyAsync(
      b_fc3_gpu[i], parameter_ + OFFSET21, 4 * sizeof(float),
      hipMemcpyHostToDevice, streams[i]
    ));
    
    CHECK_CUDA(hipStreamSynchronize(streams[i]));
  }

  // Initialize activations
  a_linear3 = new Tensor({COMPUTE_BATCH_SIZE, 4});
}

ComputeEngine::~ComputeEngine() {
  pthread_mutex_destroy(&mutex_queue);
  pthread_cond_destroy(&cond_queue);

  delete a_linear3;
}

void ComputeEngine::set_input(float *input_buf_) {
  input_buf = input_buf_;
  input_to_process = input_buf;
}

void ComputeEngine::set_output(float *output_buf_) {
  output_buf = output_buf_;
}

void ComputeEngine::run() {
  pthread_create(&thread, NULL, ComputeEngine::run_func, this);
}

void ComputeEngine::join() {
  pthread_join(thread, NULL);
}

void ComputeEngine::push(int num_input) {
  pthread_mutex_lock(&mutex_queue);
  if (num_input_ready == 0) pthread_cond_signal(&cond_queue);
  num_input_ready += num_input;
  pthread_mutex_unlock(&mutex_queue);
}

int ComputeEngine::pop() {
  int num_input = 0;
  pthread_mutex_lock(&mutex_queue);
  if (num_input_ready == 0) pthread_cond_wait(&cond_queue, &mutex_queue);
  num_input = std::min(num_input_ready, POP_BATCH_SIZE);
  num_input_ready -= num_input;
  pthread_mutex_unlock(&mutex_queue);
  return num_input;
}

void ComputeEngine::inference(int num_input) {
  DEBUG_PRINT("Inference %d\n", num_input);

  CHECK_CUDA(hipSetDevice(gpu_idx));
  
  for (int batch = 0; batch < num_input; batch+=COMPUTE_BATCH_SIZE) {
    DEBUG_PRINT("Inference %d/%d\n", num_input_processed+1, num_input);

    int now_batch_size = std::min(COMPUTE_BATCH_SIZE, num_input - batch);

    // Conv block 1 : Conv1d + LayerNorm + ReLU + MaxPool1d
    {
      CHECK_CUDA(hipMemcpyAsync(
        a_input_gpu[gpu_idx], input_to_process + batch * VOCAB_SIZE * MAX_LENGTH,
        now_batch_size * 70 * 1014 * sizeof(float),
        hipMemcpyHostToDevice, streams[gpu_idx]
      ));

      dim3 grid(CEIL_DIV(256, C1D_K7_BM), CEIL_DIV(1008, C1D_K7_BN));
      dim3 block(C1D_K7_BM * C1D_K7_BN);
      conv1d_k7_cuda<<<grid, block, 0, streams[gpu_idx]>>>(
        a_input_gpu[gpu_idx], w_conv1_gpu[gpu_idx], b_conv1_gpu[gpu_idx], a_conv1_gpu[gpu_idx],
        now_batch_size, 1008, 70, 256,
        0
      );

      layernorm_cuda<<<now_batch_size, LNORM_TPB, 0, streams[gpu_idx]>>>(
        a_conv1_gpu[gpu_idx], gamma_conv1_gpu[gpu_idx], beta_conv1_gpu[gpu_idx], a_layernorm1_gpu[gpu_idx],
        now_batch_size, 256, 1008
      );

      maxpool1d_k3_cuda<<<now_batch_size, 1008/3, 0, streams[gpu_idx]>>>(
        a_layernorm1_gpu[gpu_idx], a_pool1_gpu[gpu_idx],
        now_batch_size, 256, 1008,
        1
      );
    }

    // Conv block 2 : Conv1d + ReLU + MaxPool1d
    {
      dim3 grid(CEIL_DIV(256, C1D_K7_BM), CEIL_DIV(330, C1D_K7_BN));
      dim3 block(C1D_K7_BM * C1D_K7_BN);
      conv1d_k7_cuda<<<grid, block, 0, streams[gpu_idx]>>>(
        a_pool1_gpu[gpu_idx], w_conv2_gpu[gpu_idx], b_conv2_gpu[gpu_idx], a_conv2_gpu[gpu_idx],
        now_batch_size, 330, 256, 256,
        1
      );

      maxpool1d_k3_cuda<<<now_batch_size, 330/3, 0, streams[gpu_idx]>>>(
        a_conv2_gpu[gpu_idx], a_pool2_gpu[gpu_idx],
        now_batch_size, 256, 330,
        0
      );
    }

    // Conv block 3 : Conv1d + ReLU
    {
      dim3 grid(CEIL_DIV(256, C1D_K3_BM), CEIL_DIV(108, C1D_K3_BN));
      dim3 block(C1D_K3_BM * C1D_K3_BN);
      conv1d_k3_cuda<<<grid, block, 0, streams[gpu_idx]>>>(
        a_pool2_gpu[gpu_idx], w_conv3_gpu[gpu_idx], b_conv3_gpu[gpu_idx], a_conv3_gpu[gpu_idx],
        now_batch_size, 108, 256, 256,
        1
      );

    }

    // Conv block 4 : Conv1d + ReLU
    {

      dim3 grid(CEIL_DIV(256, C1D_K3_BM), CEIL_DIV(106, C1D_K3_BN));
      dim3 block(C1D_K3_BM * C1D_K3_BN);
      conv1d_k3_cuda<<<grid, block, 0, streams[gpu_idx]>>>(
        a_conv3_gpu[gpu_idx], w_conv4_gpu[gpu_idx], b_conv4_gpu[gpu_idx], a_conv4_gpu[gpu_idx],
        now_batch_size, 106, 256, 256,
        1
      );
    }

    // Conv block 5 : Conv1d + ReLU
    {
      dim3 grid(CEIL_DIV(256, C1D_K3_BM), CEIL_DIV(104, C1D_K3_BN));
      dim3 block(C1D_K3_BM * C1D_K3_BN);
      conv1d_k3_cuda<<<grid, block, 0, streams[gpu_idx]>>>(
        a_conv4_gpu[gpu_idx], w_conv5_gpu[gpu_idx], b_conv5_gpu[gpu_idx], a_conv5_gpu[gpu_idx],
        now_batch_size, 104, 256, 256,
        1
      );
    }


    // Conv block 6 : Conv1d + LayerNorm + ReLU + MaxPool1d
    {
      dim3 grid(CEIL_DIV(256, C1D_K3_BM), CEIL_DIV(102, C1D_K3_BN));
      dim3 block(C1D_K3_BM * C1D_K3_BN);
      conv1d_k3_cuda<<<grid, block, 0, streams[gpu_idx]>>>(
        a_conv5_gpu[gpu_idx], w_conv6_gpu[gpu_idx], b_conv6_gpu[gpu_idx], a_conv6_gpu[gpu_idx],
        now_batch_size, 102, 256, 256,
        0
      );

      layernorm_cuda<<<now_batch_size, LNORM_TPB, 0, streams[gpu_idx]>>>(
        a_conv6_gpu[gpu_idx], gamma_conv6_gpu[gpu_idx], beta_conv6_gpu[gpu_idx], a_layernorm6_gpu[gpu_idx],
        now_batch_size, 256, 102
      );

      maxpool1d_k3_cuda<<<now_batch_size, 102/3, 0, streams[gpu_idx]>>>(
        a_layernorm6_gpu[gpu_idx], a_collapse_gpu[gpu_idx],
        now_batch_size, 256, 102,
        1
      );

    }

    // FC block 1 : Linear + ReLU
    {
      dim3 grid(CEIL_DIV(now_batch_size, LIN_REG_BM), CEIL_DIV(1024, LIN_REG_BN));
      dim3 block(LIN_REG_BM, LIN_REG_BN);
      linear_reg_cuda<<<grid, block, 0, streams[gpu_idx]>>>(
        a_collapse_gpu[gpu_idx], w_fc1_gpu[gpu_idx], b_fc1_gpu[gpu_idx], a_linear1_gpu[gpu_idx],
        now_batch_size, 8704, 1024,
        1
      );
    }

    // FC block 2 : Linear + ReLU
    {
      dim3 grid(CEIL_DIV(now_batch_size, LIN_REG_BM), CEIL_DIV(1024, LIN_REG_BN));
      dim3 block(LIN_REG_BM, LIN_REG_BN);
      linear_reg_cuda<<<grid, block, 0, streams[gpu_idx]>>>(
        a_linear1_gpu[gpu_idx], w_fc2_gpu[gpu_idx], b_fc2_gpu[gpu_idx], a_linear2_gpu[gpu_idx],
        now_batch_size, 1024, 1024,
        1
      );
    }

    // FC block 3 : Linear
    {
      dim3 grid(CEIL_DIV(now_batch_size, LIN_NAIVE_BM), CEIL_DIV(4, LIN_NAIVE_BN));
      dim3 block(LIN_NAIVE_BM, LIN_NAIVE_BN);
      linear_naive_cuda<<<grid, block, 0, streams[gpu_idx]>>>(
        a_linear2_gpu[gpu_idx], w_fc3_gpu[gpu_idx], b_fc3_gpu[gpu_idx], a_linear3_gpu[gpu_idx],
        now_batch_size, 1024, 4,
        0
      );

      CHECK_CUDA(hipMemcpyAsync(
        a_linear3->buf, a_linear3_gpu[gpu_idx], now_batch_size * 4 * sizeof(float),
        hipMemcpyDeviceToHost, streams[gpu_idx]
      ));

      CHECK_CUDA(hipStreamSynchronize(streams[gpu_idx]));
    }

    int single_logit_size = a_linear3->num_elem() / now_batch_size;

    for (int b = 0; b < now_batch_size; b++) {
      float max_val = -1e99f;
      int max_idx = 0;
      for (int i = 0; i < single_logit_size; ++i) {
        if (a_linear3->buf[b * single_logit_size + i] > max_val) {
          max_val = a_linear3->buf[b * single_logit_size + i];
          max_idx = i;
        }
      }
      output_buf[num_input_processed++] = (float)max_idx;
    }

  }

  input_to_process += num_input * VOCAB_SIZE * MAX_LENGTH;
}

void *ComputeEngine::run_func(void *arg) {
  ComputeEngine *engine = (ComputeEngine *) arg;
  while (engine->num_input_processed < engine->num_input) {
    int num_input = engine->pop();
    engine->inference(num_input);
  }
  return NULL;
}


/** SECTION: Classifier interface **/

void initialize_classifier(float *parameter, int N) {
  int len_name;
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Get_processor_name(processor_name, &len_name);
  iam_root = (mpi_rank == 0);

  // Broadcast parameters
  if (parameter == nullptr)
    parameter = (float *) calloc(NUM_PARAMETER, sizeof(float));

  MPI_Bcast(parameter, NUM_PARAMETER, MPI_FLOAT, 0, MPI_COMM_WORLD);

  for (int ce_idx = 0; ce_idx < NGPU; ++ce_idx)
    compute_engines[ce_idx] = new ComputeEngine(parameter, N / mpi_size / NGPU, ce_idx);
}

void classifier_root(float *input_, float *output_, int N) {
  // Scatter input & initialize memory
  DEBUG_PRINT("Scatter input\n");
  MPI_Scatter(
    input_, N * VOCAB_SIZE * MAX_LENGTH / mpi_size, MPI_FLOAT,
    MPI_IN_PLACE, N * VOCAB_SIZE * MAX_LENGTH / mpi_size, MPI_FLOAT, 
    0, MPI_COMM_WORLD
  );
  DEBUG_PRINT("Scatter input done\n");
  
  // Compute
  for (int ce_idx = 0; ce_idx < NGPU; ++ce_idx) {
    compute_engines[ce_idx]->set_input(input_ + ce_idx * N / mpi_size / NGPU * VOCAB_SIZE * MAX_LENGTH);
    compute_engines[ce_idx]->set_output(output_ + ce_idx * N / mpi_size / NGPU);
    compute_engines[ce_idx]->run();
    compute_engines[ce_idx]->push(N / mpi_size / NGPU);
  }

  for (int ce_idx = 0; ce_idx < NGPU; ++ce_idx) {
    compute_engines[ce_idx]->join();
  }

  // Gather output
  MPI_Gather(
    MPI_IN_PLACE, N / mpi_size, MPI_FLOAT,
    output_, N / mpi_size, MPI_FLOAT, 0, MPI_COMM_WORLD
  );

}

void classifier_nonroot(float *input_, float *output_, int N) {
  // Scatter input & initialize memory
  DEBUG_PRINT("Scatter input\n");
  if (input_ == nullptr) 
    input_ = (float *) calloc(
      N * VOCAB_SIZE * MAX_LENGTH / mpi_size, 
      sizeof(float)
    );
  if (output_ == nullptr) 
    output_ = (float *) calloc(N / mpi_size, sizeof(float));

  MPI_Scatter(
    MPI_IN_PLACE, N * VOCAB_SIZE * MAX_LENGTH / mpi_size, MPI_FLOAT,
    input_, N * VOCAB_SIZE * MAX_LENGTH / mpi_size, MPI_FLOAT, 
    0, MPI_COMM_WORLD
  );
  DEBUG_PRINT("Scatter input done\n");

  // Compute
  for (int ce_idx = 0; ce_idx < NGPU; ++ce_idx) {
    compute_engines[ce_idx]->set_input(input_ + ce_idx * N / mpi_size / NGPU * VOCAB_SIZE * MAX_LENGTH);
    compute_engines[ce_idx]->set_output(output_ + ce_idx * N / mpi_size / NGPU);
    compute_engines[ce_idx]->run();
    compute_engines[ce_idx]->push(N / mpi_size / NGPU);
  }

  for (int ce_idx = 0; ce_idx < NGPU; ++ce_idx) {
    compute_engines[ce_idx]->join();
  }

  // Gather output
  MPI_Gather(
    output_, N / mpi_size, MPI_FLOAT,
    MPI_IN_PLACE, N / mpi_size, MPI_FLOAT, 0, MPI_COMM_WORLD
  );
}

void classifier(float *input_, float *output_, int N) {
  if (iam_root) classifier_root(input_, output_, N);
  else classifier_nonroot(input_, output_, N);

}

void finalize_classifier() {
  for (int ce_idx = 0; ce_idx < NGPU; ++ce_idx) {
    delete compute_engines[ce_idx];
  }
}
