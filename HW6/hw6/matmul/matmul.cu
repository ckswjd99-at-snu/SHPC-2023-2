#include "hip/hip_runtime.h"
/******************************************************************************
 *  OPTIMIZATION NOTE
 * 
 *  [OPT TARGET]
 *  Total latency under 0.11 sec
 *  
 *  [OPT LOG]
 *  Last optimization: hyperparameter tuning
 *   - increase THREAD_M from 8 to 16
 *  Baseline latency:   0.098886 sec
 * 
 *  [LATENCY BREAKDOWN]
 *  0. total latency:       0.098886 sec (100.0%)
 *  1. MPI_Scatter:         0.000000 sec (  0.0%)
 *  2. MPI_Bcast:           0.000000 sec (  0.0%)
 *  3. hipMemcpyAsync(B):  0.007043 sec (  7.1%)
 *  3. hipMemcpyAsync(A):  0.025976 sec ( 26.3%)
 *  4. matmul_kernel:       0.053739 sec ( 54.4%)
 *  5. hipMemcpyAsync(C):  0.000000 sec (  0.0%)
 *  6. MPI_Gather:          0.000000 sec (  0.0%)
 *  7. etc(error):          0.012128 sec ( 12.3%)
 * 
 *  => Plan: autotune hyperparameters
 *  
******************************************************************************/

#include "matmul.h"
#include "util.h"

#include <omp.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <pthread.h>

/** UTIL FUNCS **/

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))


/** CUDA CONSTS **/
#define NGPU 4


/** KERNEL CONSTS **/
#define BLOCK_M     128
#define BLOCK_N     128
#define BLOCK_K     8
#define THREAD_M    16
#define THREAD_N    8
#define VEC_SIZE    4
#define NUM_TM      (BLOCK_M / THREAD_M)
#define NUM_TN      (BLOCK_N / THREAD_N)
#define NUM_THS     (NUM_TM * NUM_TN)
#define LDNK_STRD   (NUM_THS / (BLOCK_K / VEC_SIZE))
#define NUM_BDIM    ((BLOCK_M * BLOCK_N) / (THREAD_M * THREAD_N))


/** GLOBALS **/
static int mpi_rank, mpi_world_size;
int M_node_start, M_node_end, M_node_size;
static int Mbegin[NGPU], Mend[NGPU];
static int ngpu;
static hipStream_t streams[NGPU];
static hipStream_t streams_mem[NGPU];
hipEvent_t htod_event[NGPU], dtoh_event[NGPU];
static float *A_gpu[NGPU], *B_gpu[NGPU], *C_gpu[NGPU];


/** FUNCS **/
static __global__ void matmul_kernel(
  float *A, float *B, float *C, int M, int N, int K
) {
  // SMEM ALLOC
  __shared__ float Asub[BLOCK_K][BLOCK_M+4];
  __shared__ float Bsub[BLOCK_K][BLOCK_N+4];

  // REG ALLOC
  float sum[THREAD_M][THREAD_N] = {0.0};
  float tempA[THREAD_M];
  float tempB[THREAD_N];

  float *A_offset = A + K * blockIdx.x * BLOCK_M;
  float *B_offset = B + blockIdx.y * BLOCK_N;
  float *C_offset = C + N * blockIdx.x * BLOCK_M + blockIdx.y * BLOCK_N;

  int tx = threadIdx.x / NUM_TN;
  int ty = threadIdx.x % NUM_TN;

  int lnk_A = threadIdx.x / (BLOCK_K / VEC_SIZE);
  int lk_A = threadIdx.x % (BLOCK_K / VEC_SIZE);

  int lnk_B = threadIdx.x / BLOCK_K;
  int lk_B = threadIdx.x % BLOCK_K;

  // ITER THROUGH K
  for (int k=0; k<K; k+=BLOCK_K) {
    for (int lda=0; lda<BLOCK_M; lda+=LDNK_STRD) {
      float4 A_offset_temp = *reinterpret_cast<float4 *>(&A_offset[K * (lda + lnk_A) + lk_A * VEC_SIZE]);
      Asub[lk_A * VEC_SIZE + 0][lda + lnk_A] = A_offset_temp.x;
      Asub[lk_A * VEC_SIZE + 1][lda + lnk_A] = A_offset_temp.y;
      Asub[lk_A * VEC_SIZE + 2][lda + lnk_A] = A_offset_temp.z;
      Asub[lk_A * VEC_SIZE + 3][lda + lnk_A] = A_offset_temp.w;
    }
    for (int ldb=0; ldb<BLOCK_N; ldb+=LDNK_STRD) {
      *reinterpret_cast<float4 *>(&Bsub[lk_B][ldb + lnk_B * VEC_SIZE])
      = *reinterpret_cast<float4 *>(&B_offset[N * lk_B + ldb + lnk_B * VEC_SIZE]);
    }
    
    __syncthreads();

    A_offset += BLOCK_K;
    B_offset += BLOCK_K * N;

    for (int bk=0; bk<BLOCK_K; bk++) {
      for (int tm=0; tm<THREAD_M; tm+=VEC_SIZE) {
        *reinterpret_cast<float4 *>(&tempA[tm])
         = *reinterpret_cast<float4 *>(&Asub[bk][THREAD_M * tx + tm]);
      }
      for (int tn=0; tn<THREAD_N; tn+=VEC_SIZE) {
        *reinterpret_cast<float4 *>(&tempB[tn])
         = *reinterpret_cast<float4 *>(&Bsub[bk][THREAD_N * ty + tn]);
      }

      for (int tm=0; tm<THREAD_M; tm++) {
        for (int tn=0; tn<THREAD_N; tn++) {
          sum[tm][tn] += tempA[tm] * tempB[tn];
        }
      }
    }

    __syncthreads();
  }

  for (int tm=0; tm<THREAD_M; tm++) {
    for (int tn=0; tn<THREAD_N; tn+=VEC_SIZE) {
      *reinterpret_cast<float4 *>(&C_offset[N * (THREAD_M * tx + tm) + THREAD_N * ty + tn])
      = *reinterpret_cast<float4 *>(&sum[tm][tn]);
    }
  }

} 

void matmul(float *A, float *B, float *C, int M, int N, int K) {

  const int NUM_WORKLOAD = 1;

  // #pragma omp parallel for num_threads(ngpu)
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipMemcpyAsync(
      B_gpu[i], B, K * N * sizeof(float),
      hipMemcpyHostToDevice, streams_mem[i]
    ));
  }

  #pragma omp parallel for num_threads(ngpu)
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));

    for (int wl=0; wl<NUM_WORKLOAD; wl++) {

      int Mbegin_wl = Mbegin[i] + wl * (Mend[i] - Mbegin[i]) / NUM_WORKLOAD;
      int Mend_wl   = Mbegin[i] + (wl + 1) * (Mend[i] - Mbegin[i]) / NUM_WORKLOAD;
      
      // Async memcpy H->D on each GPU
      CHECK_CUDA(hipMemcpyAsync(
        A_gpu[i], &A[Mbegin_wl * K],
        (Mend_wl - Mbegin_wl) * K * sizeof(float),
        hipMemcpyHostToDevice, streams_mem[i]
      ));

      CHECK_CUDA(hipEventRecord(htod_event[i], streams_mem[i]));
      CHECK_CUDA(hipStreamWaitEvent(streams[i], htod_event[i], 0));

      // Run kernels asynchronously on each GPU
      dim3 blockDim(NUM_BDIM);
      dim3 gridDim(CEIL_DIV(Mend_wl - Mbegin_wl, BLOCK_M), CEIL_DIV(N, BLOCK_N));
      matmul_kernel<<<gridDim, blockDim, 0, streams[i]>>>(
          A_gpu[i], B_gpu[i], C_gpu[i], Mend_wl - Mbegin_wl, N, K);
      CHECK_CUDA(hipGetLastError());

      CHECK_CUDA(hipEventRecord(dtoh_event[i], streams[i]));
      CHECK_CUDA(hipStreamWaitEvent(streams_mem[i], dtoh_event[i], 0));

      // Async memcpy D->H on each GPU
      CHECK_CUDA(hipMemcpyAsync(
        &C[Mbegin_wl * N], C_gpu[i],
        (Mend_wl - Mbegin_wl) * N * sizeof(float),
        hipMemcpyDeviceToHost, streams_mem[i]
      ));
    }
  }

  // Wait for all async jobs to finish
  for (int i = 0; i < ngpu; i++) {
    hipSetDevice(i);
    hipStreamSynchronize(streams[i]);
  }

}


void matmul_initialize(int M, int N, int K) {
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);

  CHECK_CUDA(hipGetDeviceCount(&ngpu));

  printf("[rank %d] Number of devices: %d\n", mpi_rank, ngpu);
  hipDeviceProp_t props[4];
  for (int i = 0; i < ngpu; ++i) {
    CHECK_CUDA(hipGetDeviceProperties(&props[i], i));
    printf("[rank %d] device %d: %s\n", mpi_rank, i, props[i].name);
  }

  M_node_start = M * mpi_rank / mpi_world_size;
  M_node_end = M * (mpi_rank + 1) / mpi_world_size;
  M_node_size = M_node_end - M_node_start;

  for (int i = 0; i < ngpu; i++) {
    Mbegin[i] = M_node_start + M_node_size * i / ngpu;
    Mend[i] = M_node_start + M_node_size * (i + 1) / ngpu;
    if (i == ngpu - 1) Mend[i] = M_node_end;
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamCreate(&streams[i]));
    CHECK_CUDA(hipStreamCreate(&streams_mem[i]));
    CHECK_CUDA(hipEventCreate(&htod_event[i]));
    CHECK_CUDA(hipEventCreate(&dtoh_event[i]));
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(
        hipMalloc(&A_gpu[i], (Mend[i] - Mbegin[i]) * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&B_gpu[i], K * N * sizeof(float)));
    CHECK_CUDA(
        hipMalloc(&C_gpu[i], (Mend[i] - Mbegin[i]) * N * sizeof(float)));
  }
}


void matmul_finalize() {
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipFree(A_gpu[i]));
    CHECK_CUDA(hipFree(B_gpu[i]));
    CHECK_CUDA(hipFree(C_gpu[i]));
    CHECK_CUDA(hipStreamDestroy(streams[i]));
    CHECK_CUDA(hipStreamDestroy(streams_mem[i]));
    CHECK_CUDA(hipEventDestroy(htod_event[i]));
    CHECK_CUDA(hipEventDestroy(dtoh_event[i]));
  }
}
