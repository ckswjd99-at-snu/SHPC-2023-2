#include "hip/hip_runtime.h"
#include "matmul.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <mpi.h>

// #define DEBUG

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

static int mpi_rank, mpi_world_size;

static __global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                                     int K) {
  int j = blockDim.x * blockIdx.x + threadIdx.x;
  int i = blockDim.y * blockIdx.y + threadIdx.y;
  if (i >= M || j >= N) return;
  float sum = 0.0;
  for (int k = 0; k < K; ++k) sum += A[i * K + k] * B[k * N + j];
  C[i * N + j] = sum;
}

#define NGPU 4

int M_node_start, M_node_end, M_node_size;
static int Mbegin[NGPU], Mend[NGPU];
static int ngpu;
static hipStream_t streams[NGPU];
static float *A_gpu[NGPU], *B_gpu[NGPU], *C_gpu[NGPU];


void matmul(const float *A, const float *B, float *C, int M, int N, int K) {

  // Scatter mat A
  float *Abuf = (float *)A;
  if (mpi_rank == 0) {
    MPI_Scatter(A, M * K / mpi_world_size, MPI_FLOAT, MPI_IN_PLACE, M * K / mpi_world_size, MPI_FLOAT, 0, MPI_COMM_WORLD);
  }
  else {
    MPI_Scatter(NULL, M * K / mpi_world_size, MPI_FLOAT, Abuf + K * M_node_start, M * K / mpi_world_size, MPI_FLOAT, 0, MPI_COMM_WORLD);
  }

  // Broadcast mat B
  float *Bbuf = (float *)B;

  MPI_Bcast(Bbuf, K * N, MPI_FLOAT, 0, MPI_COMM_WORLD);
  
  #ifdef DEBUG
  printf("[rank %d] Bbuf[0] = %f\n", mpi_rank, Bbuf[0]);
  printf("[rank %d] Bbuf[-1] = %f\n", mpi_rank, Bbuf[K*N-1]);
  #endif

  // Async memcpy H->D on each GPU
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMemcpyAsync(A_gpu[i], &A[Mbegin[i] * K],
                               (Mend[i] - Mbegin[i]) * K * sizeof(float),
                               hipMemcpyHostToDevice, streams[i]));
    CHECK_CUDA(hipMemcpyAsync(B_gpu[i], B, K * N * sizeof(float),
                               hipMemcpyHostToDevice, streams[i]));
  }

  // Run kernels asynchronously on each GPU
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    dim3 blockDim(16, 16);
    dim3 gridDim((N + 16 - 1) / 16, (Mend[i] - Mbegin[i] + 16 - 1) / 16);
    matmul_kernel<<<gridDim, blockDim, 0, streams[i]>>>(
        A_gpu[i], B_gpu[i], C_gpu[i], Mend[i] - Mbegin[i], N, K);
    CHECK_CUDA(hipGetLastError());
  }

  // Async memcpy D->H on each GPU
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMemcpyAsync(&C[Mbegin[i] * N], C_gpu[i],
                               (Mend[i] - Mbegin[i]) * N * sizeof(float),
                               hipMemcpyDeviceToHost, streams[i]));
  }

  // Wait for all async jobs to finish
  for (int i = 0; i < ngpu; i++) {
    hipSetDevice(i);
    hipStreamSynchronize(streams[i]);
  }

  for (int i=0; i<ngpu; i++) {
    printf("[rank %d] device %d: %f\n", mpi_rank, i, C[Mbegin[i] * N]);
  }

  // Gather mat C
  if (mpi_rank == 0) {
    MPI_Gather(MPI_IN_PLACE, M * N / mpi_world_size, MPI_FLOAT, C, M * N / mpi_world_size, MPI_FLOAT, 0, MPI_COMM_WORLD);
  }
  else {
    MPI_Gather(C + N * M_node_start, M * N / mpi_world_size, MPI_FLOAT, NULL, M * N / mpi_world_size, MPI_FLOAT, 0, MPI_COMM_WORLD);
  }
}


void matmul_initialize(int M, int N, int K) {
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);

  CHECK_CUDA(hipGetDeviceCount(&ngpu));

  printf("[rank %d] Number of devices: %d\n", mpi_rank, ngpu);
  hipDeviceProp_t props[4];
  for (int i = 0; i < ngpu; ++i) {
    CHECK_CUDA(hipGetDeviceProperties(&props[i], i));
    printf("[rank %d] device %d: %s\n", mpi_rank, i, props[i].name);
  }

  M_node_start = M * mpi_rank / mpi_world_size;
  M_node_end = M * (mpi_rank + 1) / mpi_world_size;
  M_node_size = M_node_end - M_node_start;

  for (int i = 0; i < ngpu; i++) {
    Mbegin[i] = M_node_start + M_node_size * i / ngpu;
    Mend[i] = M_node_start + M_node_size * (i + 1) / ngpu;
    if (i == ngpu - 1) Mend[i] = M_node_end;
  }

  #ifdef DEBUG
  for (int i = 0; i < ngpu; i++) {
    printf("[rank %d] device %d: Mbegin = %d, Mend = %d\n", mpi_rank, i, Mbegin[i], Mend[i]);
  }
  #endif

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamCreate(&streams[i]));
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(
        hipMalloc(&A_gpu[i], (Mend[i] - Mbegin[i]) * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&B_gpu[i], K * N * sizeof(float)));
    CHECK_CUDA(
        hipMalloc(&C_gpu[i], (Mend[i] - Mbegin[i]) * N * sizeof(float)));
  }
}


void matmul_finalize() {
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipFree(A_gpu[i]));
    CHECK_CUDA(hipFree(B_gpu[i]));
    CHECK_CUDA(hipFree(C_gpu[i]));
    CHECK_CUDA(hipStreamDestroy(streams[i]));
  }
}
